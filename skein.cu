#include "hip/hip_runtime.h"
/**
* SKEIN512 80 + SHA256 64
* by tpruvot@github - 2015
* Optimized by sp-hash@github - 2015
*/

extern "C" {
#include "sph/sph_skein.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include <openssl/sha.h>

static uint32_t foundnonces[MAX_GPUS][2];

extern void skein512_cpu_setBlock_80(uint32_t thr_id,void *pdata);
extern void skein512_cpu_hash_80_50(int thr_id, uint32_t threads, uint32_t startNounce, int swapu, uint64_t target, uint32_t *h_found);
extern void skein512_cpu_hash_80_52(int thr_id, uint32_t threads, uint32_t startNounce, int swapu, uint64_t target, uint32_t *h_found);

extern "C" void skeincoinhash(void *output, const void *input)
{
	sph_skein512_context ctx_skein;
	SHA256_CTX sha256;

	uint32_t hash[16];

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, input, 80);
	sph_skein512_close(&ctx_skein, hash);

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, (unsigned char *)hash, 64);
	SHA256_Final((unsigned char *)hash, &sha256);

	memcpy(output, hash, 32);
}

static __inline uint32_t swab32_if(uint32_t val, bool iftrue)
{
	return iftrue ? swab32(val) : val;
}

static bool init[MAX_GPUS] = { 0 };

int scanhash_skeincoin(int thr_id, uint32_t *pdata,
								  const uint32_t *ptarget, uint32_t max_nonce,
								  uint32_t *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	const int swap = 1;

	uint32_t intensity = (device_sm[device_map[thr_id]] > 500) ? 1 << 28 : 1 << 27;;
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity); // 256*4096
	throughput = min(throughput, max_nonce - first_nonce);

	if (opt_benchmark)
	{
		((uint64_t*)ptarget)[3] = 0x3000f0000;
	}
	uint64_t target = ((uint64_t*)ptarget)[3];
	if (!init[thr_id])
	{
		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
		if (opt_n_gputhreads == 1)
		{
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}

		cuda_check_cpu_init(thr_id, throughput);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k = 0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	skein512_cpu_setBlock_80(thr_id, (void*)endiandata);
	do
	{

		if (device_sm[device_map[thr_id]] > 500)
			skein512_cpu_hash_80_52(thr_id, throughput, pdata[19], swap, target, foundnonces[thr_id]);
		else
			skein512_cpu_hash_80_50(thr_id, throughput, pdata[19], swap, target, foundnonces[thr_id]);

		if (foundnonces[thr_id][0] != 0xffffffff)
		{
			uint32_t vhash64[8];

			endiandata[19] = swab32_if(foundnonces[thr_id][0], swap);
			
			skeincoinhash(vhash64, endiandata);

			uint64_t test = ((uint64_t*)vhash64)[3];
			if (test <= target && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput+1;

				if (opt_debug || opt_benchmark)
					applog(LOG_INFO, "GPU #%d: found nonce $%08X", thr_id, foundnonces[thr_id][0]);
				if (foundnonces[thr_id][1] != 0xffffffff)
				{
					endiandata[19] = swab32_if(foundnonces[thr_id][1], swap);
					skeincoinhash(vhash64, endiandata);
					uint64_t test2 = ((uint64_t*)vhash64)[3];
					if (test2 <= target && fulltest(vhash64, ptarget))
					{
						if (opt_debug || opt_benchmark)
							applog(LOG_INFO, "GPU #%d: found second nonce $%08X", thr_id, foundnonces[thr_id][1]);
						pdata[19 + res] = swab32_if(foundnonces[thr_id][1], !swap);
						res++;
					}
					else
					{
						if (test2 != target) applog(LOG_WARNING, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces[thr_id][1]);
					}
				}
				pdata[19] = swab32_if(foundnonces[thr_id][0], !swap);
				return res;
			}
			else 
			{
				if (test != target)
					applog(LOG_WARNING, "GPU #%d: result for nonce $%08X does not validate on CPU!", thr_id, foundnonces[thr_id][0]);
				else
					applog(LOG_WARNING, "Lost work: #%d", test);

			}
		}
		pdata[19] += throughput;
	} while (pdata[19] < max_nonce && !work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
