#include "hip/hip_runtime.h"
/**
 * Blake-256 Cuda Kernel (Tested on SM 5.0)
 *
 * Tanguy Pruvot - Nov. 2014
 */
extern "C" {
#include "sph/sph_blake.h"
}

#include "cuda_helper.h"

#include <memory.h>

static __device__ uint64_t cuda_swab32ll(uint64_t x) {
	return MAKE_ULONGLONG(cuda_swab32(_LOWORD(x)), cuda_swab32(_HIWORD(x)));
}

__constant__ static uint32_t  c_data[20];

__constant__ static uint8_t sigma[16][16];
static uint8_t  c_sigma[16][16] = {
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 }
};

static const uint32_t  c_IV256[8] = {
	0x6A09E667, 0xBB67AE85,
	0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C,
	0x1F83D9AB, 0x5BE0CD19
};

__device__ __constant__ static uint32_t cpu_h[8];

__device__ __constant__ static  uint32_t  u256[16];
static const uint32_t  c_u256[16] = {
	0x243F6A88, 0x85A308D3,
	0x13198A2E, 0x03707344,
	0xA4093822, 0x299F31D0,
	0x082EFA98, 0xEC4E6C89,
	0x452821E6, 0x38D01377,
	0xBE5466CF, 0x34E90C6C,
	0xC0AC29B7, 0xC97C50DD,
	0x3F84D5B5, 0xB5470917
};

#define GS2(a,b,c,d,x) { \
	const uint8_t idx1 = sigma[r][x]; \
	const uint8_t idx2 = sigma[r][x+1]; \
	v[a] += (m[idx1] ^ u256[idx2]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ u256[idx1]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}

#define ROTR32(x, n) (((x) >> (n)) | ((x) << (32 - (n))))
#define hostGS(a,b,c,d,x) { \
	const uint8_t idx1 = c_sigma[r][x]; \
	const uint8_t idx2 = c_sigma[r][x+1]; \
	v[a] += (m[idx1] ^ c_u256[idx2]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 16); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 12); \
\
	v[a] += (m[idx2] ^ c_u256[idx1]) + v[b]; \
	v[d] = ROTR32(v[d] ^ v[a], 8); \
	v[c] += v[d]; \
	v[b] = ROTR32(v[b] ^ v[c], 7); \
	}

#define GSPREC(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ u256[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x1032); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a],0, 0x0321); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
		}

__host__ __forceinline__
static void blake256_compress1st(uint32_t *h, const uint32_t *block, const uint32_t T0)
{
	uint32_t m[16];
	uint32_t v[16];
	
	for (int i = 0; i < 16; i++) {
		m[i] = block[i];
	}

	for (int i = 0; i < 8; i++)
		v[i] = h[i];

	v[8] = c_u256[0];
	v[9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	for (int r = 0; r < 14; r++) {
		/* column step */
		hostGS(0, 4, 0x8, 0xC, 0x0);
		hostGS(1, 5, 0x9, 0xD, 0x2);
		hostGS(2, 6, 0xA, 0xE, 0x4);
		hostGS(3, 7, 0xB, 0xF, 0x6);
		/* diagonal step */
		hostGS(0, 5, 0xA, 0xF, 0x8);
		hostGS(1, 6, 0xB, 0xC, 0xA);
		hostGS(2, 7, 0x8, 0xD, 0xC);
		hostGS(3, 4, 0x9, 0xE, 0xE);
	}

	h[0] ^= v[0] ^ v[8];
	h[1] ^= v[1] ^ v[9];
	h[2] ^= v[2] ^ v[10];
	h[3] ^= v[3] ^ v[11];
	h[4] ^= v[4] ^ v[12];
	h[5] ^= v[5] ^ v[13];
	h[6] ^= v[6] ^ v[14];
	h[7] ^= v[7] ^ v[15];
}

__device__ __forceinline__
static void blake256_compress2nd(uint32_t *h, const uint32_t *block, const uint32_t T0)
{
	uint32_t v[16];

	const uint32_t c_Padding[12] = {
		0x80000000, 0, 0, 0,
		0, 0, 0, 0,
		0, 1, 0, 640
	};

	uint32_t m[16]=
	{
		block[0], block[1], block[2], block[3],
		c_Padding[0], c_Padding[1], c_Padding[2], c_Padding[3],
		c_Padding[4], c_Padding[5], c_Padding[6], c_Padding[7],
		c_Padding[8], c_Padding[9], c_Padding[10], c_Padding[11]
	};

	#pragma unroll 8
	for (int i = 0; i < 8; i++)
		v[i] = h[i];

	v[8] =  u256[0];
	v[9] =  u256[1];
	v[10] = u256[2];
	v[11] = u256[3];
	v[12] = u256[4] ^ T0;
	v[13] = u256[5] ^ T0;
	v[14] = u256[6];
	v[15] = u256[7];

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	GSPREC(0, 4, 0x8, 0xC, 0, 1);
	GSPREC(1, 5, 0x9, 0xD, 2, 3);
	GSPREC(2, 6, 0xA, 0xE, 4, 5);
	GSPREC(3, 7, 0xB, 0xF, 6, 7);
	GSPREC(0, 5, 0xA, 0xF, 8, 9);
	GSPREC(1, 6, 0xB, 0xC, 10, 11);
	GSPREC(2, 7, 0x8, 0xD, 12, 13);
	GSPREC(3, 4, 0x9, 0xE, 14, 15);
	//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	GSPREC(0, 4, 0x8, 0xC, 14, 10);
	GSPREC(1, 5, 0x9, 0xD, 4, 8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0, 2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5, 3);
	//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5, 2);
	GSPREC(3, 7, 0xB, 0xF, 15, 13);
	GSPREC(0, 5, 0xA, 0xF, 10, 14);
	GSPREC(1, 6, 0xB, 0xC, 3, 6);
	GSPREC(2, 7, 0x8, 0xD, 7, 1);
	GSPREC(3, 4, 0x9, 0xE, 9, 4);
	//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	GSPREC(0, 4, 0x8, 0xC, 7, 9);
	GSPREC(1, 5, 0x9, 0xD, 3, 1);
	GSPREC(2, 6, 0xA, 0xE, 13, 12);
	GSPREC(3, 7, 0xB, 0xF, 11, 14);
	GSPREC(0, 5, 0xA, 0xF, 2, 6);
	GSPREC(1, 6, 0xB, 0xC, 5, 10);
	GSPREC(2, 7, 0x8, 0xD, 4, 0);
	GSPREC(3, 4, 0x9, 0xE, 15, 8);

	//	{ 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	GSPREC(0, 4, 0x8, 0xC, 9, 0);
	GSPREC(1, 5, 0x9, 0xD, 5, 7);
	GSPREC(2, 6, 0xA, 0xE, 2, 4);
	GSPREC(3, 7, 0xB, 0xF, 10, 15);
	GSPREC(0, 5, 0xA, 0xF, 14, 1);
	GSPREC(1, 6, 0xB, 0xC, 11, 12);
	GSPREC(2, 7, 0x8, 0xD, 6, 8);
	GSPREC(3, 4, 0x9, 0xE, 3, 13);
	//	{ 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	GSPREC(0, 4, 0x8, 0xC, 2, 12);
	GSPREC(1, 5, 0x9, 0xD, 6, 10);
	GSPREC(2, 6, 0xA, 0xE, 0, 11);
	GSPREC(3, 7, 0xB, 0xF, 8, 3);
	GSPREC(0, 5, 0xA, 0xF, 4, 13);
	GSPREC(1, 6, 0xB, 0xC, 7, 5);
	GSPREC(2, 7, 0x8, 0xD, 15, 14);
	GSPREC(3, 4, 0x9, 0xE, 1, 9);

	//	{ 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	GSPREC(0, 4, 0x8, 0xC, 12, 5);
	GSPREC(1, 5, 0x9, 0xD, 1, 15);
	GSPREC(2, 6, 0xA, 0xE, 14, 13);
	GSPREC(3, 7, 0xB, 0xF, 4, 10);
	GSPREC(0, 5, 0xA, 0xF, 0, 7);
	GSPREC(1, 6, 0xB, 0xC, 6, 3);
	GSPREC(2, 7, 0x8, 0xD, 9, 2);
	GSPREC(3, 4, 0x9, 0xE, 8, 11);

	//	{ 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	GSPREC(0, 4, 0x8, 0xC, 13, 11);
	GSPREC(1, 5, 0x9, 0xD, 7, 14);
	GSPREC(2, 6, 0xA, 0xE, 12, 1);
	GSPREC(3, 7, 0xB, 0xF, 3, 9);
	GSPREC(0, 5, 0xA, 0xF, 5, 0);
	GSPREC(1, 6, 0xB, 0xC, 15, 4);
	GSPREC(2, 7, 0x8, 0xD, 8, 6);
	GSPREC(3, 4, 0x9, 0xE, 2, 10);
	//	{ 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	GSPREC(0, 4, 0x8, 0xC, 6, 15);
	GSPREC(1, 5, 0x9, 0xD, 14, 9);
	GSPREC(2, 6, 0xA, 0xE, 11, 3);
	GSPREC(3, 7, 0xB, 0xF, 0, 8);
	GSPREC(0, 5, 0xA, 0xF, 12, 2);
	GSPREC(1, 6, 0xB, 0xC, 13, 7);
	GSPREC(2, 7, 0x8, 0xD, 1, 4);
	GSPREC(3, 4, 0x9, 0xE, 10, 5);
	//	{ 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	GSPREC(0, 4, 0x8, 0xC, 10, 2);
	GSPREC(1, 5, 0x9, 0xD, 8, 4);
	GSPREC(2, 6, 0xA, 0xE, 7, 6);
	GSPREC(3, 7, 0xB, 0xF, 1, 5);
	GSPREC(0, 5, 0xA, 0xF, 15, 11);
	GSPREC(1, 6, 0xB, 0xC, 9, 14);
	GSPREC(2, 7, 0x8, 0xD, 3, 12);
	GSPREC(3, 4, 0x9, 0xE, 13, 0);
	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	GSPREC(0, 4, 0x8, 0xC, 0, 1);
	GSPREC(1, 5, 0x9, 0xD, 2, 3);
	GSPREC(2, 6, 0xA, 0xE, 4, 5);
	GSPREC(3, 7, 0xB, 0xF, 6, 7);
	GSPREC(0, 5, 0xA, 0xF, 8, 9);
	GSPREC(1, 6, 0xB, 0xC, 10, 11);
	GSPREC(2, 7, 0x8, 0xD, 12, 13);
	GSPREC(3, 4, 0x9, 0xE, 14, 15);

	//	{ 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	GSPREC(0, 4, 0x8, 0xC, 14, 10);
	GSPREC(1, 5, 0x9, 0xD, 4, 8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0, 2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5, 3);

	//	{ 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5, 2);
	GSPREC(3, 7, 0xB, 0xF, 15, 13);
	GSPREC(0, 5, 0xA, 0xF, 10, 14);
	GSPREC(1, 6, 0xB, 0xC, 3, 6);
	GSPREC(2, 7, 0x8, 0xD, 7, 1);
	GSPREC(3, 4, 0x9, 0xE, 9, 4);
	//	{ 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	GSPREC(0, 4, 0x8, 0xC, 7, 9);
	GSPREC(1, 5, 0x9, 0xD, 3, 1);
	GSPREC(2, 6, 0xA, 0xE, 13, 12);
	GSPREC(3, 7, 0xB, 0xF, 11, 14);
	GSPREC(0, 5, 0xA, 0xF, 2, 6);
	GSPREC(1, 6, 0xB, 0xC, 5, 10);
	GSPREC(2, 7, 0x8, 0xD, 4, 0);
	GSPREC(3, 4, 0x9, 0xE, 15, 8);


	h[0] ^= v[0] ^ v[8];
	h[1] ^= v[1] ^ v[9];
	h[2] ^= v[2] ^ v[10];
	h[3] ^= v[3] ^ v[11];
	h[4] ^= v[4] ^ v[12];
	h[5] ^= v[5] ^ v[13];
	h[6] ^= v[6] ^ v[14];
	h[7] ^= v[7] ^ v[15];
}

__global__ __launch_bounds__(256,4)
void blake256_gpu_hash_80(const uint32_t threads, const uint32_t startNonce, uint64_t * Hash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t h[8];
		uint32_t input[4];

		#pragma unroll 8
		for (int i = 0; i<8; i++) { h[i] = cpu_h[i];}

		#pragma unroll 3
		for (int i = 0; i < 3; ++i) input[i] = c_data[16 + i];

		input[3] = nonce;
		blake256_compress2nd(h, input, 640);

        #pragma unroll
		for (int i = 0; i<4; i++) {
			Hash[i*threads + thread] = cuda_swab32ll(MAKE_ULONGLONG(h[2 * i], h[2*i+1]));
		}
	}
}

__host__
void blake256_cpu_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNonce, uint64_t *Hash)
{
	const uint32_t threadsperblock = 128;

	dim3 grid((threads + threadsperblock - 1) / threadsperblock);
	dim3 block(threadsperblock);

	blake256_gpu_hash_80 <<<grid, block>>> (threads, startNonce, Hash);
}

__host__
void blake256_cpu_setBlock_80(uint32_t *pdata)
{
	uint32_t h[8];
	uint32_t data[20];
	memcpy(data, pdata, 80);
	for (int i = 0; i<8; i++) {
		h[i] = c_IV256[i];
	}
	blake256_compress1st(h, pdata, 512);

	hipMemcpyToSymbol(HIP_SYMBOL(cpu_h), h, sizeof(h), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_data), data, sizeof(data), 0, hipMemcpyHostToDevice);
}

__host__
void blake256_cpu_init(int thr_id, uint32_t threads)
{
	hipMemcpyToSymbol(HIP_SYMBOL(u256), c_u256, sizeof(c_u256), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(sigma), c_sigma, sizeof(c_sigma), 0, hipMemcpyHostToDevice);
}
