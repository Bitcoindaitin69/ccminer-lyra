#include "hip/hip_runtime.h"


#include <stdio.h>
#include <memory.h>
#include "cuda_vector.h"
#define TPB 16
//

 
#define Nrow 4
#define Ncol 4
#if __CUDA_ARCH__ < 500 
#define vectype ulonglong4
#define u64type uint64_t
#define memshift 4
#elif __CUDA_ARCH__ == 500
#define u64type uint2
#define vectype uint28
#define memshift 3
#else 
#define u64type uint2
#define vectype uint28
#define memshift 3   
#endif 
__device__ vectype  *DMatrix;

 
static __device__ __forceinline__ void Gfunc_v35(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
{
	   
	a += b; d ^= a; d = SWAPDWORDS2(d);
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);

}

static __device__ __forceinline__ void Gfunc_v35_p1(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
{

	a += b; d ^= a; d = SWAPDWORDS2(d);
	c += d; b ^= c; b = ROR24(b);
}

static __device__ __forceinline__ void Gfunc_v35_p2(uint2 & a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);
}


static __device__ __forceinline__ void Gfunc_v35(uint64_t & a, uint64_t &b, uint64_t &c, uint64_t &d)
{

	a += b; d ^= a; d = ROTR64(d, 32);
	c += d; b ^= c; b = ROTR64(b, 24);
	a += b; d ^= a; d = ROTR64(d, 16);
	c += d; b ^= c; b = ROTR64(b, 63);

}

static __device__ __forceinline__ void Gfunc_v35_p1(uint64_t & a, uint64_t &b, uint64_t &c, uint64_t &d)
{

	a += b; d ^= a; d = ROTR64(d, 32);
	c += d; b ^= c; b = ROTR64(b, 24);
}

static __device__ __forceinline__ void Gfunc_v35_p2(uint64_t & a, uint64_t &b, uint64_t &c, uint64_t &d)
{

	a += b; d ^= a; d = ROTR64(d, 16);
	c += d; b ^= c; b = ROTR64(b, 63);
}

#define RORa(d) make_uint28(SWAPDWORDS2(d.x),SWAPDWORDS2(d.y),SWAPDWORDS2(d.z),SWAPDWORDS2(d.w))
#define RORb(d) make_uint28(ROR24(d.x),ROR24(d.y),ROR24(d.z),ROR24(d.w))
#define RORc(d) make_uint28(ROR16(d.x),ROR16(d.y),ROR16(d.z),ROR16(d.w))
#define RORd(d) make_uint28(ROR2(d.x,63),ROR2(d.y,63),ROR2(d.z,63),ROR2(d.w,63))


static __device__ __forceinline__ ulonglong4 make_vectype(const uint64_t  a, const uint64_t b, const uint64_t c, const uint64_t d)
{
	return make_ulonglong4(a, b, c, d);	
}

static __device__ __forceinline__ uint28 make_vectype(const uint2  a, const uint2 b, const uint2 c, const uint2 d)
{
	return make_uint28(a, b, c, d);
}


static __device__ __forceinline__ void Gfunc_v4(ulonglong4 & a, ulonglong4 &b, ulonglong4 &c, ulonglong4 &d)
{
#define ROR4(d,n) make_ulonglong4(ROTR64(d.x,n),ROTR64(d.y,n),ROTR64(d.z,n),ROTR64(d.w,n))
	a += b; d ^= a; d = ROR4(d, 32);
	c += d; b ^= c; b = ROR4(b, 24);
	a += b; d ^= a; d = ROR4(d, 16);
	c += d; b ^= c; b = ROR4(b, 63);
#undef ROR4
}

static __device__ __forceinline__ void Gfunc_v4(uint28 & a, uint28 &b, uint28 &c, uint28 &d)
{
#define ROR4(d,n) make_uint28(ROR2(d.x,n),ROR2(d.y,n),ROR2(d.z,n),ROR2(d.w,n))
	a += b; d ^= a; d = RORa(d);
	c += d; b ^= c; b = RORb(b);
	a += b; d ^= a; d = RORc(d);
	c += d; b ^= c; b = RORd(b);
#undef ROR4
}



static __device__ __forceinline__ void round_lyra64(uint64_t* s)  
{  
	Gfunc_v35(s[0], s[4], s[8], s[12]);  
	Gfunc_v35(s[1], s[5], s[9], s[13]);  
	Gfunc_v35(s[2], s[6], s[10], s[14]);  
	Gfunc_v35(s[3], s[7], s[11], s[15]);  
	Gfunc_v35(s[0], s[5], s[10], s[15]);   
	Gfunc_v35(s[1], s[6], s[11], s[12]);  
	Gfunc_v35(s[2], s[7], s[8], s[13]);  
	Gfunc_v35(s[3], s[4], s[9], s[14]);  
}

static __device__ __forceinline__ void round_lyra_v35(uint2_16* s)
{
	    Gfunc_v35(s[0].s0, s[0].s4, s[0].s8, s[0].sc);  
		Gfunc_v35(s[0].s1, s[0].s5, s[0].s9, s[0].sd);  
		Gfunc_v35(s[0].s2, s[0].s6, s[0].sa, s[0].se);  
		Gfunc_v35(s[0].s3, s[0].s7, s[0].sb, s[0].sf);  
		Gfunc_v35(s[0].s0, s[0].s5, s[0].sa, s[0].sf);  
		Gfunc_v35(s[0].s1, s[0].s6, s[0].sb, s[0].sc);  
		Gfunc_v35(s[0].s2, s[0].s7, s[0].s8, s[0].sd);  
		Gfunc_v35(s[0].s3, s[0].s4, s[0].s9, s[0].se);  
}

static __device__ __forceinline__ void round_lyra_v35(uint2* s)
{
	Gfunc_v35(s[0], s[4], s[8], s[12]);
	Gfunc_v35(s[1], s[5], s[9], s[13]);
	Gfunc_v35(s[2], s[6], s[10], s[14]);
	Gfunc_v35(s[3], s[7], s[11], s[15]);
	Gfunc_v35(s[0], s[5], s[10], s[15]);
	Gfunc_v35(s[1], s[6], s[11], s[12]);
	Gfunc_v35(s[2], s[7], s[8], s[13]);
	Gfunc_v35(s[3], s[4], s[9], s[14]);
}

static __device__ __forceinline__ void round_lyra_v35(vectype* s)
{

	Gfunc_v35(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc_v35(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc_v35(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc_v35(s[0].w, s[1].w, s[2].w, s[3].w);

	Gfunc_v35(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc_v35(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc_v35(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc_v35(s[0].w, s[1].x, s[2].y, s[3].z);

}
 
static __device__ __forceinline__ void reduceDuplex(vectype state[4], uint32_t thread)
{


	    vectype state1[3]; 
		uint32_t ps1 = (Nrow * Ncol * memshift * thread);
		uint32_t ps2 = (memshift * (Ncol-1) + memshift * Ncol + Nrow * Ncol * memshift * thread);

#pragma unroll 4
	for (int i = 0; i < Ncol; i++)
	{
        uint32_t s1 = ps1 + i*memshift;
        uint32_t s2 = ps2 - i*memshift;  
		
		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix+s1)[j]); 
 
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];
		round_lyra_v35(state); 
		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];

		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];

	}

}

static __device__ __forceinline__ void reduceDuplexV3(vectype state[4], uint32_t thread)
{


	vectype state1[3];
	uint32_t ps1 = (Nrow * Ncol * memshift * thread);
//                     colomn             row
	uint32_t ps2 = (memshift * (Ncol - 1) * Nrow + memshift * 1 + Nrow * Ncol * memshift * thread);

#pragma unroll 4
	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + Nrow * i *memshift;
		uint32_t s2 = ps2 - Nrow * i *memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);

		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];
		round_lyra_v35(state);

		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];


		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state1[j];

	}

}

static __device__ __forceinline__ void reduceDuplexRowSetupV2(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{


		vectype state2[3],state1[3];

		uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
		uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
		uint32_t ps3 = (memshift * (Ncol-1) + memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);


//#pragma unroll 1
	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 + i*memshift;
		uint32_t s3 = ps3 - i*memshift;

		for (int j = 0; j < 3; j++) 
			state1[j]= __ldg4(&(DMatrix + s1)[j]);
		for (int j = 0; j < 3; j++)
			state2[j]= __ldg4(&(DMatrix + s2)[j]);
		for (int j = 0; j < 3; j++) {
			vectype tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}
		

		round_lyra_v35(state);

		for (int j = 0; j < 3; j++) {
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}
 
		   ((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++) 
			((uint2*)state2)[j+1] ^= ((uint2*)state)[j];



		for (int j = 0; j < 3; j++)
		    (DMatrix + s2)[j] = state2[j];
		
	}


}

static __device__ __forceinline__ void reduceDuplexRowSetupV3(const int rowIn, const int rowInOut, const int rowOut, vectype state[4], uint32_t thread)
{


	vectype state2[3], state1[3];
	
	uint32_t ps1 = (memshift *  rowIn                     + Nrow * Ncol * memshift * thread);
	uint32_t ps2 = (memshift * rowInOut                   + Nrow * Ncol * memshift * thread);
	uint32_t ps3 = (Nrow * memshift * (Ncol - 1) + memshift *  rowOut + Nrow * Ncol * memshift * thread);

	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + Nrow*i*memshift;
		uint32_t s2 = ps2 + Nrow*i*memshift;
		uint32_t s3 = ps3 - Nrow*i*memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1 )[j]);
		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2 )[j]);
		for (int j = 0; j < 3; j++) {
			vectype tmp = state1[j] + state2[j];
			state[j] ^= tmp;
		}


		round_lyra_v35(state);

		for (int j = 0; j < 3; j++) {
			state1[j] ^= state[j];
			(DMatrix + s3)[j] = state1[j];
		}

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];



		for (int j = 0; j < 3; j++)
			(DMatrix + s2)[j] = state2[j];

	}


}


static __device__ __forceinline__ void reduceDuplexRowtV2(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{

		vectype state1[3],state2[3];
		uint32_t ps1 = (memshift * Ncol * rowIn + Nrow * Ncol * memshift * thread);
		uint32_t ps2 = (memshift * Ncol * rowInOut + Nrow * Ncol * memshift * thread);
		uint32_t ps3 = (memshift * Ncol * rowOut + Nrow * Ncol * memshift * thread);

//#pragma unroll 1
	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + i*memshift;
		uint32_t s2 = ps2 + i*memshift;
		uint32_t s3 = ps3 + i*memshift;


		for (int j = 0; j < 3; j++)  
			state1[j] = __ldg4(&(DMatrix + s1)[j]);


		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);


		for (int j = 0; j < 3; j++)
			          state1[j] += state2[j];

		for (int j = 0; j < 3; j++)
			          state[j] ^= state1[j];


		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
		((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

if (rowInOut != rowOut) {

	for (int j = 0; j < 3; j++)
		(DMatrix + s2)[j] = state2[j];

	for (int j = 0; j < 3; j++)
		(DMatrix + s3)[j] ^= state[j];

} else {

	for (int j = 0; j < 3; j++)
		state2[j] ^= state[j];

	for (int j = 0; j < 3; j++)
		(DMatrix + s2)[j]=state2[j];
}






	}
}

static __device__ __forceinline__ void reduceDuplexRowtV3(const int rowIn, const int rowInOut, const int rowOut, vectype* state, uint32_t thread)
{

	vectype state1[3], state2[3];
	uint32_t ps1 = (memshift * rowIn + Nrow * Ncol * memshift * thread);
	uint32_t ps2 = (memshift * rowInOut + Nrow * Ncol * memshift * thread);
	uint32_t ps3 = (memshift * rowOut + Nrow * Ncol * memshift * thread);

#pragma nounroll 
	for (int i = 0; i < Ncol; i++)
	{
		uint32_t s1 = ps1 + Nrow * i*memshift;
		uint32_t s2 = ps2 + Nrow * i*memshift;
		uint32_t s3 = ps3 + Nrow * i*memshift;


		for (int j = 0; j < 3; j++)
			state1[j] = __ldg4(&(DMatrix + s1)[j]);


		for (int j = 0; j < 3; j++)
			state2[j] = __ldg4(&(DMatrix + s2)[j]);


		for (int j = 0; j < 3; j++)
			state1[j] += state2[j];

		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];


		round_lyra_v35(state);

		((uint2*)state2)[0] ^= ((uint2*)state)[11];
		for (int j = 0; j < 11; j++)
			((uint2*)state2)[j + 1] ^= ((uint2*)state)[j];

		if (rowInOut != rowOut) {

			for (int j = 0; j < 3; j++)
				(DMatrix + s2)[j] = state2[j];

			for (int j = 0; j < 3; j++)
				(DMatrix + s3)[j] ^= state[j];

		}
		else {

			for (int j = 0; j < 3; j++)
				state2[j] ^= state[j];

			for (int j = 0; j < 3; j++)
				(DMatrix + s2)[j] = state2[j];
		}






	}
}


#if __CUDA_ARCH__ < 500
__global__	__launch_bounds__(128, 1)
#elif __CUDA_ARCH__ == 500
__global__	__launch_bounds__(16, 1)
#else
__global__	__launch_bounds__(TPB, 1)
#endif
void lyra2v2_gpu_hash_32_v3(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	vectype state[4];


	uint28 blake2b_IV[2];
	uint28 padding[2];
	if (threadIdx.x == 0) {

		((uint2_8*)blake2b_IV)[0] = {
			{ 0xf3bcc908, 0x6a09e667 },
			{ 0x84caa73b, 0xbb67ae85 },
			{ 0xfe94f82b, 0x3c6ef372 },
			{ 0x5f1d36f1, 0xa54ff53a },
			{ 0xade682d1, 0x510e527f },
			{ 0x2b3e6c1f, 0x9b05688c },
			{ 0xfb41bd6b, 0x1f83d9ab },
			{ 0x137e2179, 0x5be0cd19 }
		};
		((uint2_8*)padding)[0] = {
			{ 0x20, 0x0 },
			{ 0x20, 0x0 },
			{ 0x20, 0x0 },
			{ 0x01, 0x0 },
			{ 0x04, 0x0 },
			{ 0x04, 0x0 },
			{ 0x80, 0x0 },
			{ 0x0, 0x01000000 }
		};

	}

#if __CUDA_ARCH__ == 350
	if (thread < threads)
#endif
	{

		((uint2*)state)[0] = __ldg(&outputHash[thread]);
		((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);
		state[1] = state[0];
		state[2] = shuffle4(((vectype*)blake2b_IV)[0], 0);
		state[3] = shuffle4(((vectype*)blake2b_IV)[1], 0);

		for (int i = 0; i<12; i++)
			round_lyra_v35(state);
		state[0] ^= shuffle4(((vectype*)padding)[0], 0);
		state[1] ^= shuffle4(((vectype*)padding)[1], 0);


		for (int i = 0; i<12; i++)
			round_lyra_v35(state);

		uint32_t ps1 = (4 * memshift * 3 + 16 * memshift * thread);

		//#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			uint32_t s1 = ps1 - 4 * memshift * i;
			for (int j = 0; j < 3; j++)
				(DMatrix + s1)[j] = (state)[j];

			round_lyra_v35(state);
		}

		reduceDuplexV3(state, thread);
		reduceDuplexRowSetupV3(1, 0, 2, state, thread);
		reduceDuplexRowSetupV3(2, 1, 3, state, thread);

		uint32_t rowa;
		int prev = 3;
		for (int i = 0; i < 4; i++)
		{
			rowa = ((uint2*)state)[0].x & 3;  reduceDuplexRowtV3(prev, rowa, i, state, thread);
			prev = i;
		}

		uint32_t shift = (memshift * rowa + 16 * memshift * thread);

		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
			round_lyra_v35(state);


		outputHash[thread] = ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2];
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
		//		((vectype*)outputHash)[thread] = state[0];

	} //thread
}



#if __CUDA_ARCH__ < 500
__global__	__launch_bounds__(64, 1)
#elif __CUDA_ARCH__ == 500
__global__	__launch_bounds__(32, 1)
#else
__global__	__launch_bounds__(TPB, 1)
#endif
void lyra2v2_gpu_hash_32(uint32_t threads, uint32_t startNounce, uint2 *outputHash)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	   vectype state[4];

	   uint28 blake2b_IV[2];
	   uint28 padding[2];
	   if (threadIdx.x == 0) {

		   ((uint2_8*)blake2b_IV)[0] = {
			   { 0xf3bcc908, 0x6a09e667 },
			   { 0x84caa73b, 0xbb67ae85 },
			   { 0xfe94f82b, 0x3c6ef372 },
			   { 0x5f1d36f1, 0xa54ff53a },
			   { 0xade682d1, 0x510e527f },
			   { 0x2b3e6c1f, 0x9b05688c },
			   { 0xfb41bd6b, 0x1f83d9ab },
			   { 0x137e2179, 0x5be0cd19 }
		   };
		   ((uint2_8*)padding)[0] = {
			   { 0x20, 0x0 },
			   { 0x20, 0x0 },
			   { 0x20, 0x0 },
			   { 0x01, 0x0 },
			   { 0x04, 0x0 },
			   { 0x04, 0x0 },
			   { 0x80, 0x0 },
			   { 0x0, 0x01000000 }
		   };
       }

#if __CUDA_ARCH__ == 350
	if (thread < threads)
#endif
	{
 
		 ((uint2*)state)[0] = __ldg(&outputHash[thread]);
		 ((uint2*)state)[1] = __ldg(&outputHash[thread + threads]);
		 ((uint2*)state)[2] = __ldg(&outputHash[thread + 2 * threads]);
		 ((uint2*)state)[3] = __ldg(&outputHash[thread + 3 * threads]);

		 state[1] = state[0];

		 state[2] = shuffle4(((vectype*)blake2b_IV)[0], 0);
		 state[3] = shuffle4(((vectype*)blake2b_IV)[1], 0);

		 for (int i = 0; i<12; i++)
			 round_lyra_v35(state);
		 state[0] ^= shuffle4(((vectype*)padding)[0], 0);
		 state[1] ^= shuffle4(((vectype*)padding)[1], 0);

		 for (int i = 0; i<12; i++)
			 round_lyra_v35(state);

		uint32_t ps1 = (memshift * (Ncol - 1) + Nrow * Ncol * memshift * thread);

		for (int i = 0; i < Ncol; i++)
		{
			uint32_t s1 = ps1 - memshift * i;
			for (int j = 0; j < 3; j++)
			    (DMatrix + s1)[j] = (state)[j];

			round_lyra_v35(state);
		}


		reduceDuplex(state, thread);

		reduceDuplexRowSetupV2(1, 0, 2, state,  thread);
		reduceDuplexRowSetupV2(2, 1, 3, state,  thread);
uint32_t rowa;
int prev=3;

         for (int i = 0; i < 4; i++)
        {
	     rowa = ((uint2*)state)[0].x & 3;  reduceDuplexRowtV2(prev, rowa, i, state, thread);
         prev=i;
        }


		uint32_t shift = (memshift * Ncol * rowa + Nrow * Ncol * memshift * thread);

		for (int j = 0; j < 3; j++)
			state[j] ^= __ldg4(&(DMatrix + shift)[j]);

		for (int i = 0; i < 12; i++)
        			round_lyra_v35(state);
		

		outputHash[thread]=            ((uint2*)state)[0];
		outputHash[thread + threads] = ((uint2*)state)[1];
		outputHash[thread + 2 * threads] = ((uint2*)state)[2]; 
		outputHash[thread + 3 * threads] = ((uint2*)state)[3];
//		((vectype*)outputHash)[thread] = state[0];

	} //thread
}


__host__
void lyra2v2_cpu_init(int thr_id, uint32_t threads,uint64_t *hash)
{
	hipMemcpyToSymbol(HIP_SYMBOL(DMatrix), &hash, sizeof(hash), 0, hipMemcpyHostToDevice);
}



__host__ 
void lyra2v2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_outputHash)
{
uint32_t tpb;
	if (device_sm[device_map[thr_id]]<500) 
      tpb = 64;
	else if (device_sm[device_map[thr_id]]==500)
      tpb = 32; 
    else 
      tpb = TPB;
	dim3 grid((threads + tpb - 1) / tpb);
	dim3 block(tpb);

	if (device_sm[device_map[thr_id]] >= 500)
		lyra2v2_gpu_hash_32 << <grid, block >> > (threads, startNounce, (uint2*)d_outputHash);
    else 
    	lyra2v2_gpu_hash_32_v3 <<<grid, block>>> (threads, startNounce,(uint2*) d_outputHash);

	//MyStreamSynchronize(NULL, order, thr_id);
}

  